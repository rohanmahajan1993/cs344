#include "hip/hip_runtime.h"
#include "reference_calc.cpp"
#include "utils.h"

void your_histogram_and_prefixsum(const float* const d_logLuminance,
                                  unsigned int* const d_cdf,
                                  float &min_logLum,
                                  float &max_logLum,
                                  const size_t numRows,
                                  const size_t numCols,
                                  const size_t numBins)
{
  //TODO
  /*Here are the steps you need to implement
    1) find the minimum and maximum value in the input logLuminance channel
       store in min_logLum and max_logLum
    2) subtract them to find the range
    3) generate a histogram of all the values in the logLuminance channel using
       the formula: bin = (lum[i] - lumMin) / lumRange * numBins
    4) Perform an exclusive scan (prefix sum) on the histogram to get
       the cumulative distribution of luminance values (this should go in the
       incoming d_cdf pointer which already has been allocated for you)       */
    int threadLimit = 512;
    int blockWidth = trunc(sqrt(threadLimit));
    int blockHeight = trunc(sqrt(threadLimit));
    int numBlockRows = numRows / blockWidth + (numRows % blockWidth != 0);
    int numBlockCols = numCols / blockHeight + (numCols % blockHeight != 0);
    const dim3 blockSize(blockWidth, blockHeight, 1);
    const dim3 gridSize(numBlockRows, numBlockCols, 1);
    float *h_minValue;
    float *h_maxValue;
    float *d_maxValue;
    float *d_minValue;
    checkCudaErrors(hipMalloc(&d_minValue,   sizeof(float)));
    checkCudaErrors(hipMalloc(&d_maxValue,   sizeof(float)));
    find_optimum<<<gridSize, blockSize>>>(d_input,  d_minValue, true);
    find_optimum<<<gridSize, blockSize>>>(d_input,  d_maxValue, false);
    checkCudaErrors(hipMemcpy(d_minValue, h_minValue, sizeof(float), hipMemcpyDeviceToHost));
    checkCudaErrors(hipMemcpy(d_maxValue, h_maxValue, sizeof(float), hipMemcpyDeviceToHost));
    float difference = h_maxValue - h_minValue;
    hipDeviceSynchronize(); checkCudaErrors(hipGetLastError());
}
//This kernel takes in three color channels and recombines them
//into one image.  The alpha channel is set to 255 to represent
//that this image has no transparency.
__global__
void findOptimum(const float* const d_input, float * optimalValue, boolean isMinimum)
{
  int rowIndex = threadIdx.x + blockDim.x * blockIdx.x;
  int colIndex = threadIdx.y + blockDim.y * blockIdx.y;
  int index = (rowIndex * numCols) + colIndex;
  if (rowIndex < numRows && colIndex < numCols) {
    unsigned char red   = redChannel[index];
    unsigned char green = greenChannel[index];
    unsigned char blue  = blueChannel[index];
    //Alpha should be 255 for no transparency
    uchar4 outputPixel = make_uchar4(red, green, blue, 255);
    outputImageRGBA[index] = outputPixel;
  }
}

void cleanup() {
  checkCudaErrors(hipFree(d_red));
  checkCudaErrors(hipFree(d_green));
  checkCudaErrors(hipFree(d_blue));
  checkCudaErrors(hipFree(d_filter));
}
